#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : md5.cu
 Author      : xdegtyarev
 Version     :
 Copyright   : alexander degtyarev
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/ipc.h>
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <limits.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

// CUDA kernel that performs op on chunk of data
__global__ void baseChunkOpKernel(char* data, int chunkSize) {
	long threadSubchunkSize = chunkSize/blockDim.y;
	long threadStartOffset = threadSubchunkSize*threadIdx.y;
}

__global__ void printFileChunkOpKernel(char* data, int chunkSize) {
	long threadSubchunkSize = chunkSize/blockDim.y;
	long threadStartOffset = threadSubchunkSize*threadIdx.y;
	char* subbuf = (char*)malloc(threadSubchunkSize);
	for(int i = 0; i<threadSubchunkSize; i++){
		subbuf[i] = *(data+i+threadStartOffset);
	}
	printf("#################\n\n%d:[%s]\n\n##############",threadIdx.y,subbuf);
	free(subbuf);
}

__global__ void fileChunkCharCounterKernel(char* data, int chunkSize, unsigned long* res) {
	long threadSubchunkSize = chunkSize/blockDim.y;
	long threadStartOffset = threadSubchunkSize*threadIdx.y;
	char curr;
	for(int i = 0; i<threadSubchunkSize; i++){
		curr = *(data+i+threadStartOffset);
		res[curr]++;
	}
}

__global__ void fileChunkCompressorOpKernel(char* data, int chunkSize, char* res) {
	long threadSubchunkSize = chunkSize/blockDim.y;
	long threadStartOffset = threadSubchunkSize*threadIdx.y;
	char curr;
	char prev = 0;
	char acc = 0;
	long resC = threadStartOffset;
	for(int i = 0; i<threadSubchunkSize; i++){
		curr = *(data+i+threadStartOffset);
		if(curr == prev){
			acc++;
		}else{
			if(acc!=0){
				res[resC] = acc;
				resC++;
			}
			res[resC] = curr;
			acc = 0;
		}
	}
}

void printDeviceInfo(){
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf(" Device %d: \"%s\" with Compute %d.%d capability\n",
				   i, prop.name, prop.major, prop.minor);
		printf("Total global memory:           %lu\n", prop.totalGlobalMem);
		printf("Number of multiprocessors:     %d\n", prop.multiProcessorCount);
		printf("Number of compute cores:     %d\n", prop.multiProcessorCount*192);
		printf("Total amount of shared memory per block: %lu\n",prop.sharedMemPerBlock);
		printf("Total registers per block:     %d\n", prop.regsPerBlock);
		printf("Warp size:                     %d\n", prop.warpSize);
		printf("Maximum memory pitch:          %lu\n", prop.memPitch);
		printf("Total amount of constant memory:         %lu\n",   prop.totalConstMem);
		printf("Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("Peak Memory Bandwidth (GB/s): %f\n\n",
			   2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	  }
}

void gpuBaseFileChunkRunner(char* data, size_t len, size_t chunkSz)
{
	size_t chunksCount = 512;
	size_t devBufferSize = chunkSz*chunksCount;
	printf("TotalBytes: %lu | Device BufferSize: %lu \n",len,devBufferSize);
	char* devBuf;
	CUDA_CHECK_RETURN(hipMalloc(&devBuf, sizeof(char) * devBufferSize));

	size_t hostBytesLeft = len;
	size_t chunkSize;

	int numBlocks = 1;
	dim3 dimBlock(numBlocks,chunksCount);

	do
	{
		chunkSize = ( hostBytesLeft < devBufferSize) ? hostBytesLeft : devBufferSize;
		printf("Bytes left: %lu| total: %lu| reading: %lu\n",hostBytesLeft,len,chunkSize);
		CUDA_CHECK_RETURN(hipMemcpy(devBuf, data, chunkSize * sizeof( char ) , hipMemcpyHostToDevice));
		baseChunkOpKernel<<<numBlocks,dimBlock>>>(devBuf, chunkSize);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		data = data + chunkSize;
		hostBytesLeft = hostBytesLeft - chunkSize;
	} while( hostBytesLeft > 0 );
	CUDA_CHECK_RETURN(hipFree(devBuf));
}

void gpuFileChunkRunner(char* data, size_t len, size_t chunkSz)
{
	size_t chunksCount = 512;
	size_t devBufferSize = chunkSz*chunksCount;
	printf("TotalBytes: %lu | Device BufferSize: %lu \n",len,devBufferSize);
	char* devBuf;
	unsigned long* res;
	unsigned long* result = (unsigned long*)malloc(sizeof(unsigned long) * CHAR_MAX);
	unsigned long* accresult = (unsigned long*)malloc(sizeof(unsigned long) * CHAR_MAX);
	for(int c = 0; c<CHAR_MAX; ++c){
		result[c] = 0;
		accresult[c] = 0;
	}

	CUDA_CHECK_RETURN(hipMalloc(&devBuf, sizeof(char) * devBufferSize));
	CUDA_CHECK_RETURN(hipMalloc(&res, sizeof(unsigned long) * CHAR_MAX));

	size_t hostBytesLeft = len;
	size_t chunkSize;

	int numBlocks = 1;
	dim3 dimBlock(numBlocks,chunksCount);

	do
	{
		chunkSize = ( hostBytesLeft < devBufferSize) ? hostBytesLeft : devBufferSize;
		printf("Bytes left: %lu| total: %lu| reading: %lu\n",hostBytesLeft,len,chunkSize);
		CUDA_CHECK_RETURN(hipMemcpy(res,result,sizeof(unsigned long) * CHAR_MAX,hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(devBuf, data, chunkSize * sizeof( char ) , hipMemcpyHostToDevice));
	    fileChunkCharCounterKernel<<<numBlocks,dimBlock>>>(devBuf, chunkSize,res);
	    CUDA_CHECK_RETURN(hipDeviceSynchronize());
	    CUDA_CHECK_RETURN(hipMemcpy( result, res, sizeof(unsigned long) * CHAR_MAX, hipMemcpyDeviceToHost));

	    for(int c = 0; c<CHAR_MAX; ++c){
	    	accresult[c]=accresult[c]+result[c];
	    	result[c] = 0;
	    }
	    data = data + chunkSize;
	    hostBytesLeft = hostBytesLeft - chunkSize;
	} while( hostBytesLeft > 0 );

	printf("______\n");
	for(int c = 32; c<CHAR_MAX; ++c){
		printf("[%x] %c:%u\n",c,c,accresult[c]);
	}
	printf("______\n");
	CUDA_CHECK_RETURN(hipFree(devBuf));
	CUDA_CHECK_RETURN(hipFree(res));
	free(result);
	free(accresult);
}

int main(int argc, char *argv[])
{
	//check args
    if (argc < 2) {
    	fprintf(stderr, "%s\n", "File not specified");
        exit(EXIT_FAILURE);
    }else{
		bool debug_print = argc > 2;

		if(debug_print){
			printDeviceInfo();
		}

    	//check file
        if((ftok(argv[1],'R')) < 0){
            perror(argv[1]);
            exit(EXIT_FAILURE);
        }

        struct stat64 st;
        stat64(argv[1], &st);
        long flen = st.st_size;
    	int fd;
    	if((fd = open(argv[1],O_RDONLY))<0){
    		perror("open");
    		exit(EXIT_FAILURE);
    	}
    	void* filep;
    	if((filep = mmap (0, flen, PROT_READ, MAP_SHARED, fd, 0)) == MAP_FAILED){
            perror("MMAP fail");
    	    exit(EXIT_FAILURE);
    	}

    	gpuBaseFileChunkRunner((char*)filep,flen,512);
//    	gpuFileChunkRunner((char*)filep,flen,512);

    	if ((close (fd)) < 0) {
			perror ("close");
			return 1;
    	}

    	if ((munmap (filep, flen)) < 0) {
			perror ("munmap");
			return 1;
		}

	    return 0;
    }
}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	printf("%s returned %s (err) %d at %s line: %d \n",statement,hipGetErrorString(err),err,file,line);
	exit (1);
}

