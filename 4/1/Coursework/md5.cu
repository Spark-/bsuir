#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : md5.cu
 Author      : xdegtyarev
 Version     :
 Copyright   : alexander degtyarev
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/ipc.h>
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <limits.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

// CUDA kernel that performs op on chunk of data

__global__ void printFileChunkOpKernel(char* data, int chunkSize) {
	long threadSubchunkSize = chunkSize/blockDim.y;
	long threadStartOffset = threadSubchunkSize*threadIdx.y;
	char* subbuf = (char*)malloc(threadSubchunkSize);
	for(int i = 0; i<threadSubchunkSize; i++){
		subbuf[i] = *(data+i+threadStartOffset);
	}
	printf("#################\n\n%d:[%s]\n\n##############",threadIdx.y,subbuf);
	free(subbuf);
}

__global__ void fileChunkCharCounterKernel(char* data, int chunkSize, unsigned* res) {
	long threadSubchunkSize = chunkSize/blockDim.y;
	long threadStartOffset = threadSubchunkSize*threadIdx.y;
	for(int i = 0; i<threadSubchunkSize; i++){
		if(*(data+i+threadStartOffset)<128){
			res[*(data+i+threadStartOffset)]++;
		}
	}
}

__global__ void fileChunkCompressorOpKernel(char* data, int chunkSize) {
	long threadSubchunkSize = chunkSize/blockDim.y;
	long threadStartOffset = threadSubchunkSize*threadIdx.y;
	char curr = data[threadStartOffset];
	char* comp = (char*)malloc(threadSubchunkSize);

	for(int i = 0; i<threadSubchunkSize; i++){
		curr = *(data+i+threadStartOffset);
	}
//	for(int i = 0; i<threadSubchunkSize; i++){
//		subbuf[i] = *(data+i+threadStartOffset);
//	}
//	printf("#################\n\n%d:[%s]\n\n##############",threadIdx.y,subbuf);
}

void printDeviceInfo(){
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf(" Device %d: \"%s\" with Compute %d.%d capability\n",
				   i, prop.name, prop.major, prop.minor);
		printf("Total global memory:           %u\n", prop.totalGlobalMem);
		printf("Number of multiprocessors:     %d\n", prop.multiProcessorCount);
		printf("Number of compute cores:     %d\n", prop.multiProcessorCount*192);
		printf("Total amount of shared memory per block: %u\n",prop.sharedMemPerBlock);
		printf("Total registers per block:     %d\n", prop.regsPerBlock);
		printf("Warp size:                     %d\n", prop.warpSize);
		printf("Maximum memory pitch:          %u\n", prop.memPitch);
		printf("Total amount of constant memory:         %u\n",   prop.totalConstMem);
		printf("Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("Peak Memory Bandwidth (GB/s): %f\n\n",
			   2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	  }
}

void gpuFileChunkRunner(char* data, long len, long chunkSz)
{
	long chunksCount = 512;
	long devBufferSize = chunkSz*chunksCount;

	char* devBuf;
	unsigned* res;
	unsigned* result = (unsigned*)malloc(sizeof(unsigned) * CHAR_MAX);
	for(int c = 0; c<CHAR_MAX; ++c){
		result[c] = 0;
	}
	hipMalloc(&devBuf, sizeof(char) * devBufferSize);
	hipMalloc(&res, sizeof(unsigned) * CHAR_MAX);

	hipMemcpy(&res,result,sizeof(unsigned) * 128,hipMemcpyHostToDevice);

	char* poffset = data;
	long hostBytesLeft    = len;
	long chunkSize;

	printf("Len %ld devBufferSize %ld \n",len,devBufferSize);

	int numBlocks = 1;
	dim3 dimBlock(numBlocks,chunksCount);

	do
	{
		chunkSize = ( hostBytesLeft < devBufferSize) ? hostBytesLeft : devBufferSize;
		printf("Bytes %ld:%ld\n",hostBytesLeft,len);
	    hipMemcpy( devBuf, poffset, chunkSize * sizeof( char ) , hipMemcpyHostToDevice);
	    fileChunkCharCounterKernel<<<numBlocks,dimBlock>>>(devBuf, chunkSize,res);
	    poffset   = poffset + chunkSize;
	    hostBytesLeft = hostBytesLeft - chunkSize;
	    hipDeviceSynchronize();
	} while( hostBytesLeft > 0 );

	hipMemcpy( result, res, sizeof(unsigned) * CHAR_MAX, hipMemcpyDeviceToHost);

	printf("______\n");
	for(int c = 20; c<CHAR_MAX; ++c){
		printf("[%x] %c:%ld\n",c,c,result[c]);
	}
	printf("______\n");
	hipFree(devBuf);
	hipFree(res);
	free(result);
}

int main(int argc, char *argv[])
{
	//check args
    if (argc < 2) {
    	fprintf(stderr, "%s\n", "File not specified");
        exit(EXIT_FAILURE);
    }else{
		bool debug_print = argc > 2;

		if(debug_print){
			printDeviceInfo();
		}

    	//check file
        if((ftok(argv[1],'R')) < 0){
            perror(argv[1]);
            exit(EXIT_FAILURE);
        }

        struct stat64 st;
        stat64(argv[1], &st);
        long flen = st.st_size;
    	int fd;
    	if((fd = open(argv[1],O_RDONLY))<0){
    		perror("open");
    		exit(EXIT_FAILURE);
    	}
    	void* filep;
    	if((filep = mmap (0, flen, PROT_READ, MAP_SHARED, fd, 0)) == MAP_FAILED){
            perror("MMAP fail");
    	    exit(EXIT_FAILURE);
    	}

    	gpuFileChunkRunner((char*)filep,flen,512);

    	if ((close (fd)) < 0) {
			perror ("close");
			return 1;
    	}

    	if ((munmap (filep, flen)) < 0) {
			perror ("munmap");
			return 1;
		}

	    return 0;
    }
}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	printf("%s returned %s (err) %d at %s line: %d \n",statement,hipGetErrorString(err),err,file,line);
	exit (1);
}

